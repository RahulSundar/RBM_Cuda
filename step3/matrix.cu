#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdlib.h> 
#include "consts.h"
#include "matrix.h"

#define TILE_WIDTH 40

//-----------------------------------------------


__global__ void MatrixMult(int m, int n, int k, double *a, double *b, double *c)
{

 int row = threadIdx.y + blockIdx.y*blockDim.y;  
 int col = threadIdx.x + blockIdx.x*blockDim.x;  
 
 if((row < m) && (col < k))
 {
  double temp = 0.0;
  for (int i = 0; i < n; ++i)
  {
   temp += a[row*n+i]*b[col+i*k];
  }
  c[row*k+col] = temp; 
 }

}

//--------------------------------------------------

// Compute C = A * B
__global__ void matrixMultiplySharedMem(double * A, double * B, double * C,
  		       int numARows, int numAColumns,
			       int numBRows, int numBColumns,
			       int numCRows, int numCColumns) {
    //@@ Insert code to implement matrix multiplication here
    __shared__ double ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ double ds_N[TILE_WIDTH][TILE_WIDTH];
    int bx = blockIdx.x, by = blockIdx.y,
       tx = threadIdx.x, ty = threadIdx.y,
       Row = by * TILE_WIDTH + ty,
       Col = bx * TILE_WIDTH + tx;
    double Pvalue = 0;

    for (int m = 0; m < (numAColumns-1)/TILE_WIDTH+1; ++m) {
       if (Row < numARows && m*TILE_WIDTH+tx < numAColumns)
          ds_M[ty][tx] = A[Row*numAColumns + m*TILE_WIDTH+tx];
       else
          ds_M[ty][tx] = 0;
       if (Col < numBColumns && m*TILE_WIDTH+ty < numBRows)
          ds_N[ty][tx] = B[(m*TILE_WIDTH+ty)*numBColumns+Col];
       else
          ds_N[ty][tx] = 0;

       __syncthreads();
       for (int k = 0; k < TILE_WIDTH; ++k)
          Pvalue += ds_M[ty][k] * ds_N[k][tx];
       __syncthreads();
    }
    if (Row < numCRows && Col < numCColumns)
       C[Row*numCColumns+Col] = Pvalue;
}

//-------------------------------------------------

void matmul_gpu(int n1, int n2, int n3, double *A, double *B, double *C)
{

 double *dev_a, *dev_b, *dev_c;
 
 dim3 dimGrid((n3-1)/TILE_WIDTH+1,(n1-1)/TILE_WIDTH+1,1);
 dim3 dimBlock(TILE_WIDTH,TILE_WIDTH,1);


 hipMalloc((void**)&dev_a, n1*n2*sizeof(double));
 hipMalloc((void**)&dev_b, n2*n3*sizeof(double));
 hipMalloc((void**)&dev_c, n1*n3*sizeof(double));

 hipMemcpy(dev_a, A, n1*n2*sizeof(double), hipMemcpyHostToDevice);
 hipMemcpy(dev_b, B, n2*n3*sizeof(double), hipMemcpyHostToDevice);

 // global memory version
 //MatrixMult<<<dimGrid,dimBlock>>>(n1,n2,n3,dev_a,dev_b,dev_c);

 // shared memory version
 matrixMultiplySharedMem<<<dimGrid,dimBlock>>>(dev_a,dev_b,dev_c,n1,n2,n2,n3,n1,n3);

 hipMemcpy(C, dev_c, n1*n3*sizeof(double), hipMemcpyDeviceToHost);


 hipFree(dev_a);
 hipFree(dev_b);
 hipFree(dev_c);

}


//----------------------------------------------

